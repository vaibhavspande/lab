#include "hip/hip_runtime.h"
#include <stdio.h>
// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    int size = 1000000;             // Size of the vectors
    int *a, *b, *c;             // Host vectors
    int *dev_a, *dev_b, *dev_c; // Device vectors
    // Allocate memory for host vectors
    a = (int *)malloc(size * sizeof(int));
    b = (int *)malloc(size * sizeof(int));
    c = (int *)malloc(size * sizeof(int));
    // Initialize host vectors
    for (int i = 0; i < size; i++)
    {
        a[i] = i;
        b[i] = 2 * i;
    }
    // Allocate memory on the device for device vectors
    hipMalloc( (void **)&dev_a, size * sizeof(int));
    hipMalloc((void **)&dev_b, size * sizeof(int));
    hipMalloc((void **)&dev_c, size * sizeof(int));
    // Copy host vectors to device
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    // Launch kernel for vector addition
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    // Copy result from device to host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    clock_t start = clock();
    for (int i = 0; i < size; ++i)
    {
        c[i] = a[i] + b[i];
    }
    clock_t end = clock();
    double sequentialTime = ((double)(end - start)) / CLOCKS_PER_SEC;

    // Measure performance of parallel algorithm
    hipEvent_t parallelStart, parallelEnd;
    hipEventCreate(&parallelStart);
    hipEventCreate(&parallelEnd);
    hipEventRecord(parallelStart);

    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    hipEventRecord(parallelEnd);
    hipEventSynchronize(parallelEnd);
    float parallelTime = 0;
    hipEventElapsedTime(&parallelTime, parallelStart, parallelEnd);

    parallelTime /= 1000; // Convert milliseconds to seconds



    printf("Sequential Time: %f seconds\n", sequentialTime);
    printf("Parallel Time: %f seconds\n", parallelTime);


    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
